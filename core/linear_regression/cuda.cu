#include "hip/hip_runtime.h"
#ifndef __NVCC__
#error Not using NVCC, perhaps not available
#endif // !__NVCC__

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <numeric>
#include <algorithm>
#include <functional>
#include <iostream>
#include <iomanip>
#include <stdio.h>
#include <array>
#include "util/csv.h"

static const int ARR_SIZE = 100000;
static const int BLOCK_SIZE = 1024;

struct Coordinate
{
    double x, y;

    Coordinate(double a, double b)
    {
        x = a;
        y = b;
    }
};

enum TargetCoordinateOperation
{
    SumX,
    SumY,
    XTimesX,
    XTimesY,
    XMinusXMean,
    YMinusYMean,
    YMYMSquare,
    XMXMTimesYMYM
};

__global__ void
operateKernel(const Coordinate *coord, double *vec, double mean, TargetCoordinateOperation op)
{
    int i = threadIdx.x;
    if (op == XMinusXMean)
        vec[i] = coord[i].x - mean;
    else if (op == YMinusYMean)
        vec[i] = coord[i].y - mean;
    else if (op == YMYMSquare)
        vec[i] = (coord[i].y - mean) * 2;
    else if (op == XMXMTimesYMYM)
        vec[i] = (coord[i].x - mean) * (coord[i].y - mean);
}

hipError_t operateWithCuda(const Coordinate *coordinates, double *vector, unsigned int size, double mean, TargetCoordinateOperation op)
{
    Coordinate *dev_coord;
    double *dev_vec;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate GPU memory for one input and one output
    cudaStatus = hipMalloc((void **)&dev_coord, size * sizeof(Coordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_vec, size * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coordinates, size * sizeof(Coordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    // Launch a kernel on the GPU with one thread for each element
    // operateKernel<<<1, size>>>(dev_coord, dev_vec, mean, op);
    operateKernel<<<1, size>>>(dev_coord, dev_vec, mean, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from GPU to CPU
    cudaStatus = hipMemcpy(vector, dev_vec, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    goto Clean;

Clean:
    hipFree(dev_coord);
    hipFree(dev_vec);

    return cudaStatus;
}

__global__ void
pushBackKernel(const Coordinate *coord, double *vec, TargetCoordinateOperation op)
{
    int i = threadIdx.x;
    if (op == SumX)
        vec[i] = coord[i].x;
    else if (op == SumY)
        vec[i] = coord[i].y;
    else if (op == XTimesY)
        vec[i] = coord[i].x * coord[i].y;
    else if (op == XTimesX)
        vec[i] = coord[i].x * coord[i].x;
}

hipError_t pushBackWithCuda(const Coordinate *coordinates, double *vector, unsigned int size, TargetCoordinateOperation op)
{
    Coordinate *dev_coord;
    double *dev_vec;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate GPU memory for one input and one output
    cudaStatus = hipMalloc((void **)&dev_coord, size * sizeof(Coordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_vec, size * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coordinates, size * sizeof(Coordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    // Launch a kernel on the GPU with one thread for each element
    pushBackKernel<<<1, size>>>(dev_coord, dev_vec, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from GPU to CPU
    cudaStatus = hipMemcpy(vector, dev_vec, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    goto Clean;

Clean:
    hipFree(dev_coord);
    hipFree(dev_vec);

    return cudaStatus;
}

double accumulate(double sum, int n, const double vec[])
{
    for (int i = 0; i < n; i++)
    {
        sum += vec[i];
    }
    return sum;
}

double CalculateGradient(const std::vector<Coordinate> &coordinates)
{
    // auto timer = Timer("CalculateGradient");

    int n = coordinates.size();
    const Coordinate *coord = &coordinates[0];

    // Sum of x*y
    double vec_xy[ARR_SIZE], sum_xy = 0;
    pushBackWithCuda(coord, vec_xy, n, XTimesY);
    sum_xy = accumulate(sum_xy, n, vec_xy);

    // Sum of x
    double vec_x[ARR_SIZE], sum_x = 0;
    pushBackWithCuda(coord, vec_x, n, SumX);
    sum_x = accumulate(sum_x, n, vec_x);

    // Sum of y
    double vec_y[ARR_SIZE], sum_y = 0;
    pushBackWithCuda(coord, vec_y, n, SumY);
    sum_x = accumulate(sum_y, n, vec_y);

    // Sum of x*x
    double vec_xx[ARR_SIZE], sum_xx = 0;
    pushBackWithCuda(coord, vec_xx, n, XTimesX);
    sum_xx = accumulate(sum_xx, n, vec_xx);

    return (n * sum_xy - sum_x * sum_y) / (n * sum_xx - sum_x * sum_x);
}

double CalculateYIntercept(const std::vector<Coordinate> &coordinates, const double gradient)
{
    int n = coordinates.size();
    const Coordinate *coord = &coordinates[0];

    // Sum of y
    double vec_y[ARR_SIZE], sum_y = 0;
    pushBackWithCuda(coord, vec_y, n, SumY);
    sum_y = accumulate(sum_y, n, vec_y);

    // Sum of x
    double vec_x[ARR_SIZE], sum_x = 0;
    pushBackWithCuda(coord, vec_x, n, SumX);
    sum_x = accumulate(sum_x, n, vec_x);

    return (1 / (double)n) * (sum_y - gradient * sum_x);
}

enum Operation
{
    SumOfX1,
    SumOfX2,
    SumOfY,
    SumOfSquaresX1,
    SumOfSquaresX2,
    SumOfProductsX1Y,
    SumOfProductsX2Y,
    SumofProductsX1X2
};

struct MultivariateCoordinate
{
    double xs[2];
    double y;

    MultivariateCoordinate(double x1, double x2, double y)
    {
        this->xs[0] = x1;
        this->xs[1] = x2;
        this->y = y;
    }
};

__global__ void calculate_sum(const MultivariateCoordinate *coord, double *out, Operation op)
{
    int idx = threadIdx.x;
    double sum = 0;

    switch (op)
    {
    case SumOfX1:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += coord[i].xs[0];
        break;
    case SumOfX2:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += coord[i].xs[1];
        break;
    case SumOfY:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += coord[i].y;
        break;
    default:
        break;
    }

    __shared__ double r[BLOCK_SIZE];
    r[idx] = sum;
    __syncthreads();
    for (int size = BLOCK_SIZE / 2; size > 0; size /= 2)
    { // uniform
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

__global__ void calculate_sum(const MultivariateCoordinate *coord, double *out, double mean, Operation op)
{
    int idx = threadIdx.x;
    double sum = 0;

    switch (op)
    {
    case SumOfSquaresX1:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[0] - mean) * (coord[i].xs[0] - mean);
        break;
    case SumOfSquaresX2:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[1] - mean) * (coord[i].xs[1] - mean);
        break;
    default:
        break;
    }

    __shared__ double r[BLOCK_SIZE];
    r[idx] = sum;
    __syncthreads();
    for (int size = BLOCK_SIZE / 2; size > 0; size /= 2)
    { // uniform
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

__global__ void calculate_sum(const MultivariateCoordinate *coord, double *out, double mean1, double mean2, Operation op)
{
    int idx = threadIdx.x;
    double sum = 0;

    switch (op)
    {
    case SumOfProductsX1Y:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[0] - mean1) * (coord[i].y - mean2);
        break;
    case SumOfProductsX2Y:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[1] - mean1) * (coord[i].y - mean2);
        break;
    case SumofProductsX1X2:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[0] - mean1) * (coord[i].xs[1] - mean2);
        break;
    default:
        break;
    }

    __shared__ double r[BLOCK_SIZE];
    r[idx] = sum;
    __syncthreads();
    for (int size = BLOCK_SIZE / 2; size > 0; size /= 2)
    { // uniform
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

double SumWithCuda(const std::vector<MultivariateCoordinate> &mc, Operation op)
{
    // Variables on CPU
    const MultivariateCoordinate *coord = &mc[0];
    double out[ARR_SIZE];
    int n = mc.size();

    // Variables on GPU
    MultivariateCoordinate *dev_coord;
    double *dev_out;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate memory on GPU
    cudaStatus = hipMalloc((void **)&dev_coord, n * sizeof(MultivariateCoordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_out, n * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coord, n * sizeof(MultivariateCoordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy input failed!");
        goto Clean;
    }

    // Launch kernel
    calculate_sum<<<1, BLOCK_SIZE>>>(dev_coord, dev_out, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from output to input
    cudaStatus = hipMemcpy(out, dev_out, n * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy output failed!");
        goto Clean;
    }

    return out[0];

Clean:
    hipFree(dev_coord);
    hipFree(dev_out);
}

double SumWithCuda(const std::vector<MultivariateCoordinate> &mc, double mean, Operation op)
{
    // Variables on CPU
    const MultivariateCoordinate *coord = &mc[0];
    double out[ARR_SIZE];
    int n = mc.size();

    // Variables on GPU
    MultivariateCoordinate *dev_coord;
    double *dev_out;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate memory on GPU
    cudaStatus = hipMalloc((void **)&dev_coord, n * sizeof(MultivariateCoordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_out, n * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coord, n * sizeof(MultivariateCoordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy input failed!");
        goto Clean;
    }

    // Launch kernel
    calculate_sum<<<1, BLOCK_SIZE>>>(dev_coord, dev_out, mean, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from output to input
    cudaStatus = hipMemcpy(out, dev_out, n * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy output failed!");
        goto Clean;
    }

    return out[0];

Clean:
    hipFree(dev_coord);
    hipFree(dev_out);
}

double SumWithCuda(const std::vector<MultivariateCoordinate> &mc, double mean1, double mean2, Operation op)
{
    // Variables on CPU
    const MultivariateCoordinate *coord = &mc[0];
    double out[ARR_SIZE];
    int n = mc.size();

    // Variables on GPU
    MultivariateCoordinate *dev_coord;
    double *dev_out;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate memory on GPU
    cudaStatus = hipMalloc((void **)&dev_coord, n * sizeof(MultivariateCoordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_out, n * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coord, n * sizeof(MultivariateCoordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy input failed!");
        goto Clean;
    }

    // Launch kernel
    calculate_sum<<<1, BLOCK_SIZE>>>(dev_coord, dev_out, mean1, mean2, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from output to input
    cudaStatus = hipMemcpy(out, dev_out, n * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy output failed!");
        goto Clean;
    }

    return out[0];

Clean:
    hipFree(dev_coord);
    hipFree(dev_out);
}

std::tuple<std::pair<double, double>, double> CalculateGradientAndYIntercept(const std::vector<MultivariateCoordinate> &mc)
{
    Operation op;

    // Total number of coordinates
    int n = mc.size();

    // Calculate sum of x1, sum of x2 and sum of y
    op = SumOfX1;
    double sum_of_x1 = SumWithCuda(mc, op);
    op = SumOfX2;
    double sum_of_x2 = SumWithCuda(mc, op);
    op = SumOfY;
    double sum_of_y = SumWithCuda(mc, op);

    // Calculate mean
    double mean_x1 = sum_of_x1 / n;
    double mean_x2 = sum_of_x2 / n;
    double mean_y = sum_of_y / n;

    // Calculate sum of squares
    op = SumOfSquaresX1;
    double sum_of_squares_x1 = SumWithCuda(mc, mean_x1, op);
    op = SumOfSquaresX2;
    double sum_of_squares_x2 = SumWithCuda(mc, mean_x2, op);

    // Calculate sum of products
    op = SumOfProductsX1Y;
    double sum_of_products_x1_y = SumWithCuda(mc, mean_x1, mean_y, op);
    op = SumOfProductsX2Y;
    double sum_of_products_x2_y = SumWithCuda(mc, mean_x2, mean_y, op);
    op = SumofProductsX1X2;
    double sum_of_products_x1_x2 = SumWithCuda(mc, mean_x1, mean_x2, op);

    // Calculate b1, b2 and a
    double b1 = (sum_of_products_x1_y * sum_of_squares_x2 - sum_of_products_x1_x2 * sum_of_products_x2_y) / (sum_of_squares_x1 * sum_of_squares_x2 - sum_of_products_x1_x2 * sum_of_products_x1_x2);

    double b2 = (sum_of_products_x2_y * sum_of_squares_x1 - sum_of_products_x1_x2 * sum_of_products_x1_y) / (sum_of_squares_x1 * sum_of_squares_x2 - sum_of_products_x1_x2 * sum_of_products_x1_x2);

    double a = mean_y - (b1 * mean_x1) - (b2 * mean_x2);

    // Test return
    return std::make_tuple(std::make_pair(b1, b2), a);
}

// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
void EndCuda()
{
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}

std::vector<MultivariateCoordinate> read_mock_csv(const char *filepath, int *num_of_rows)
{
    io::CSVReader<5> in(filepath);
    in.read_header(io::ignore_extra_column, "name", "points", "skill", "assists", "salary");
    std::string name, skill;
    double points, assists, salary;
    std::vector<MultivariateCoordinate> coord;

    if (num_of_rows == nullptr)
        // read all rows
        while (in.read_row(name, points, skill, assists, salary))
            coord.push_back(MultivariateCoordinate(points, assists, salary));
    else
        // read specified number of rows
        for (int i = 0; i < *num_of_rows; i++)
        {
            in.read_row(name, points, skill, assists, salary);
            coord.push_back(MultivariateCoordinate(points, assists, salary));
        }

    return coord;
}

double PredictY(std::array<double, 2> Xs, double b1, double b2, double a)
{
    return (Xs[0] * b1) + (Xs[1] * b2) - a;
}

int main(int argc, char **argv)
{
    int num_of_rows = -1;

    // Check for user arguments
    if (argc != 2 && argc != 1)
    {
        fprintf(stderr, "Usage: ./cuda [num_of_rows]\n");
        return 1;
    }

    if (argv[1] != nullptr)
        num_of_rows = std::stoi(argv[1]);

    std::vector<MultivariateCoordinate> data = read_mock_csv("dataset/mock.csv", num_of_rows == -1 ? nullptr : &num_of_rows);

    // print how many coordinates in red
    std::cout << "\033[1;31m[DEBUG] " << data.size() << " coordinate is loaded\033[0m" << std::endl;

    double b1 = 0.0, b2 = 0.0, a = 0.0;

    auto start = std::chrono::high_resolution_clock::now();

    std::forward_as_tuple(std::tie(b1, b2), a) = CalculateGradientAndYIntercept(data);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;

    const char *operation_name = "InitModel";
    float ms = duration.count() * 1000.0f;
    const char *to_print = operation_name == nullptr ? "An unknown operation" : operation_name;
    std::cout << "\033[1;31m[TIMER] " << to_print << " took " << ms << "ms\033[0m\n";

    // print the straight line formula
    std::cout << "\nThe following is the calculated formula of straight line: " << std::endl;
    std::cout << "y = " + std::to_string(b1) + "x₁ + " + std::to_string(b2) + "x₂ + " + std::to_string(a) << std::endl;

    // get user input
    std::array<double, 2> Xs;
    std::cout << "\nEnter the X values separated by space, for example: 32.3 8.9" << std::endl;
    std::cin >> Xs[0] >> Xs[1];

    // calculate the predicted Y value and print it
    double predicted = PredictY(Xs, b1, b2, a);
    std::cout << std::fixed << std::setprecision(4);
    std::cout << "\nThe predicted Y value is " << predicted << " (4 d.p.)" << std::endl;

    EndCuda();
}