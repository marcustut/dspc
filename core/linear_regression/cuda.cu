#include "hip/hip_runtime.h"
#ifndef __NVCC__
#error Not using NVCC, perhaps not available
#endif // !__NVCC__

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <numeric>
#include <algorithm>
#include <stdio.h>

const unsigned int ARR_SIZE = 1024;

struct Coordinate
{
    double x, y;

    Coordinate(double a, double b)
    {
        x = a;
        y = b;
    }
};

enum TargetCoordinateOperation
{
    SumX,
    SumY,
    XTimesX,
    XTimesY,
    XMinusXMean,
    YMinusYMean,
    YMYMSquare,
    XMXMTimesYMYM
};

__global__ void
operateKernel(const Coordinate *coord, double *vec, double mean, TargetCoordinateOperation op)
{
    int i = threadIdx.x;
    if (op == XMinusXMean)
        vec[i] = coord[i].x - mean;
    else if (op == YMinusYMean)
        vec[i] = coord[i].y - mean;
    else if (op == YMYMSquare)
        vec[i] = (coord[i].y - mean) * 2;
    else if (op == XMXMTimesYMYM)
        vec[i] = (coord[i].x - mean) * (coord[i].y - mean);
}

hipError_t operateWithCuda(const Coordinate *coordinates, double *vector, unsigned int size, double mean, TargetCoordinateOperation op)
{
    Coordinate *dev_coord;
    double *dev_vec;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate GPU memory for one input and one output
    cudaStatus = hipMalloc((void **)&dev_coord, size * sizeof(Coordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_vec, size * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coordinates, size * sizeof(Coordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    // Launch a kernel on the GPU with one thread for each element
    operateKernel<<<1, size>>>(dev_coord, dev_vec, mean, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from GPU to CPU
    cudaStatus = hipMemcpy(vector, dev_vec, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    goto Clean;

Clean:
    hipFree(dev_coord);
    hipFree(dev_vec);

    return cudaStatus;
}

__global__ void
pushBackKernel(const Coordinate *coord, double *vec, TargetCoordinateOperation op)
{
    int i = threadIdx.x;
    if (op == SumX)
        vec[i] = coord[i].x;
    else if (op = SumY)
        vec[i] = coord[i].y;
    else if (op == XTimesY)
        vec[i] = coord[i].x * coord[i].y;
    else if (op == XTimesX)
        vec[i] = coord[i].x * coord[i].x;
}

hipError_t pushBackWithCuda(const Coordinate *coordinates, double *vector, unsigned int size, TargetCoordinateOperation op)
{
    Coordinate *dev_coord;
    double *dev_vec;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate GPU memory for one input and one output
    cudaStatus = hipMalloc((void **)&dev_coord, size * sizeof(Coordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_vec, size * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coordinates, size * sizeof(Coordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    // Launch a kernel on the GPU with one thread for each element
    pushBackKernel<<<1, size>>>(dev_coord, dev_vec, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from GPU to CPU
    cudaStatus = hipMemcpy(vector, dev_vec, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    goto Clean;

Clean:
    hipFree(dev_coord);
    hipFree(dev_vec);

    return cudaStatus;
}

double accumulate(double sum, int n, const double vec[])
{
    for (int i = 0; i < n; i++)
    {
        sum += vec[i];
    }
    return sum;
}

double CalculateGradient(const std::vector<Coordinate> &coordinates)
{
    // auto timer = Timer("CalculateGradient");

    int n = coordinates.size();
    const Coordinate *coord = &coordinates[0];

    // Sum of x*y
    double vec_xy[ARR_SIZE], sum_xy = 0;
    pushBackWithCuda(coord, vec_xy, n, XTimesY);
    sum_xy = accumulate(sum_xy, n, vec_xy);

    // Sum of x
    double vec_x[ARR_SIZE], sum_x = 0;
    pushBackWithCuda(coord, vec_x, n, SumX);
    sum_x = accumulate(sum_x, n, vec_x);

    // Sum of y
    double vec_y[ARR_SIZE], sum_y = 0;
    pushBackWithCuda(coord, vec_y, n, SumY);
    sum_x = accumulate(sum_y, n, vec_y);

    // Sum of x*x
    double vec_xx[ARR_SIZE], sum_xx = 0;
    pushBackWithCuda(coord, vec_xx, n, XTimesX);
    sum_xx = accumulate(sum_xx, n, vec_xx);

    return (n * sum_xy - sum_x * sum_y) / (n * sum_xx - sum_x * sum_x);
}

double CalculateMultiGradient(const std::vector<Coordinate> &coordinates)
{
    int n = coordinates.size();
    const Coordinate *coord = &coordinates[0];

    // Calculate mean_x
    double vec_x[ARR_SIZE];
    pushBackWithCuda(coord, vec_x, n, SumX);
    double sum_x = accumulate(sum_x, n, vec_x);
    double mean_x = sum_x / n;

    // Calculate mean_y
    double vec_y[ARR_SIZE];
    pushBackWithCuda(coord, vec_y, n, SumY);
    double sum_y = accumulate(sum_y, n, vec_y);
    double mean_y = sum_y / n;

    // ???????? mind block
}

double CalculateYIntercept(const std::vector<Coordinate> &coordinates, const double gradient)
{
    int n = coordinates.size();
    const Coordinate *coord = &coordinates[0];

    // Sum of y
    double vec_y[ARR_SIZE], sum_y = 0;
    pushBackWithCuda(coord, vec_y, n, SumY);
    sum_y = accumulate(sum_y, n, vec_y);

    // Sum of x
    double vec_x[ARR_SIZE], sum_x = 0;
    pushBackWithCuda(coord, vec_x, n, SumX);
    sum_x = accumulate(sum_x, n, vec_x);

    return (1 / (double)n) * (sum_y - gradient * sum_x);
}

// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
void EndCuda()
{
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}