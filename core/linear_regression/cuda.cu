#include "hip/hip_runtime.h"
#ifndef __NVCC__
#error Not using NVCC, perhaps not available
#endif // !__NVCC__

#ifndef __HIPCC__
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include <vector>
#include <numeric>
#include <algorithm>
#include <functional>
#include <iostream>
#include <stdio.h>
#include <array>

static const int ARR_SIZE = 100000;
static const int BLOCK_SIZE = 1024;

struct Coordinate
{
    double x, y;

    Coordinate(double a, double b)
    {
        x = a;
        y = b;
    }
};

enum TargetCoordinateOperation
{
    SumX,
    SumY,
    XTimesX,
    XTimesY,
    XMinusXMean,
    YMinusYMean,
    YMYMSquare,
    XMXMTimesYMYM
};

__global__ void
operateKernel(const Coordinate *coord, double *vec, double mean, TargetCoordinateOperation op)
{
    int i = threadIdx.x;
    if (op == XMinusXMean)
        vec[i] = coord[i].x - mean;
    else if (op == YMinusYMean)
        vec[i] = coord[i].y - mean;
    else if (op == YMYMSquare)
        vec[i] = (coord[i].y - mean) * 2;
    else if (op == XMXMTimesYMYM)
        vec[i] = (coord[i].x - mean) * (coord[i].y - mean);
}

hipError_t operateWithCuda(const Coordinate *coordinates, double *vector, unsigned int size, double mean, TargetCoordinateOperation op)
{
    Coordinate *dev_coord;
    double *dev_vec;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate GPU memory for one input and one output
    cudaStatus = hipMalloc((void **)&dev_coord, size * sizeof(Coordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_vec, size * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coordinates, size * sizeof(Coordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    // Launch a kernel on the GPU with one thread for each element
    // operateKernel<<<1, size>>>(dev_coord, dev_vec, mean, op);
    operateKernel<<<1, size>>>(dev_coord, dev_vec, mean, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from GPU to CPU
    cudaStatus = hipMemcpy(vector, dev_vec, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    goto Clean;

Clean:
    hipFree(dev_coord);
    hipFree(dev_vec);

    return cudaStatus;
}

__global__ void
pushBackKernel(const Coordinate *coord, double *vec, TargetCoordinateOperation op)
{
    int i = threadIdx.x;
    if (op == SumX)
        vec[i] = coord[i].x;
    else if (op = SumY)
        vec[i] = coord[i].y;
    else if (op == XTimesY)
        vec[i] = coord[i].x * coord[i].y;
    else if (op == XTimesX)
        vec[i] = coord[i].x * coord[i].x;
}

hipError_t pushBackWithCuda(const Coordinate *coordinates, double *vector, unsigned int size, TargetCoordinateOperation op)
{
    Coordinate *dev_coord;
    double *dev_vec;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate GPU memory for one input and one output
    cudaStatus = hipMalloc((void **)&dev_coord, size * sizeof(Coordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_vec, size * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coordinates, size * sizeof(Coordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    // Launch a kernel on the GPU with one thread for each element
    pushBackKernel<<<1, size>>>(dev_coord, dev_vec, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from GPU to CPU
    cudaStatus = hipMemcpy(vector, dev_vec, size * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed!");
        goto Clean;
    }

    goto Clean;

Clean:
    hipFree(dev_coord);
    hipFree(dev_vec);

    return cudaStatus;
}

double accumulate(double sum, int n, const double vec[])
{
    for (int i = 0; i < n; i++)
    {
        sum += vec[i];
    }
    return sum;
}

double CalculateGradient(const std::vector<Coordinate> &coordinates)
{
    // auto timer = Timer("CalculateGradient");

    int n = coordinates.size();
    const Coordinate *coord = &coordinates[0];

    // Sum of x*y
    double vec_xy[ARR_SIZE], sum_xy = 0;
    pushBackWithCuda(coord, vec_xy, n, XTimesY);
    sum_xy = accumulate(sum_xy, n, vec_xy);

    // Sum of x
    double vec_x[ARR_SIZE], sum_x = 0;
    pushBackWithCuda(coord, vec_x, n, SumX);
    sum_x = accumulate(sum_x, n, vec_x);

    // Sum of y
    double vec_y[ARR_SIZE], sum_y = 0;
    pushBackWithCuda(coord, vec_y, n, SumY);
    sum_x = accumulate(sum_y, n, vec_y);

    // Sum of x*x
    double vec_xx[ARR_SIZE], sum_xx = 0;
    pushBackWithCuda(coord, vec_xx, n, XTimesX);
    sum_xx = accumulate(sum_xx, n, vec_xx);

    return (n * sum_xy - sum_x * sum_y) / (n * sum_xx - sum_x * sum_x);
}

double CalculateYIntercept(const std::vector<Coordinate> &coordinates, const double gradient)
{
    int n = coordinates.size();
    const Coordinate *coord = &coordinates[0];

    // Sum of y
    double vec_y[ARR_SIZE], sum_y = 0;
    pushBackWithCuda(coord, vec_y, n, SumY);
    sum_y = accumulate(sum_y, n, vec_y);

    // Sum of x
    double vec_x[ARR_SIZE], sum_x = 0;
    pushBackWithCuda(coord, vec_x, n, SumX);
    sum_x = accumulate(sum_x, n, vec_x);

    return (1 / (double)n) * (sum_y - gradient * sum_x);
}

enum Operation
{
    SumOfX1,
    SumOfX2,
    SumOfY,
    SumOfSquaresX1,
    SumOfSquaresX2,
    SumOfProductsX1Y,
    SumOfProductsX2Y,
    SumofProductsX1X2
};

struct MultivariateCoordinate
{
    double xs[2];
    double y;

    MultivariateCoordinate(double x1, double x2, double y)
    {
        this->xs[0] = x1;
        this->xs[1] = x2;
        this->y = y;
    }
};

__global__ void calculate_sum(const MultivariateCoordinate *coord, double *out, Operation op)
{
    int idx = threadIdx.x;
    double sum = 0;

    switch (op)
    {
    case SumOfX1:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += coord[i].xs[0];
        break;
    case SumOfX2:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += coord[i].xs[1];
        break;
    case SumOfY:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += coord[i].y;
        break;
    }

    __shared__ double r[BLOCK_SIZE];
    r[idx] = sum;
    __syncthreads();
    for (int size = BLOCK_SIZE / 2; size > 0; size /= 2)
    { // uniform
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

__global__ void calculate_sum(const MultivariateCoordinate *coord, double *out, double mean, Operation op)
{
    int idx = threadIdx.x;
    double sum = 0;

    switch (op)
    {
    case SumOfSquaresX1:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[0] - mean) * (coord[i].xs[0] - mean);
        break;
    case SumOfSquaresX2:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[1] - mean) * (coord[i].xs[1] - mean);
        break;
    }

    __shared__ double r[BLOCK_SIZE];
    r[idx] = sum;
    __syncthreads();
    for (int size = BLOCK_SIZE / 2; size > 0; size /= 2)
    { // uniform
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

__global__ void calculate_sum(const MultivariateCoordinate *coord, double *out, double mean1, double mean2, Operation op)
{
    int idx = threadIdx.x;
    double sum = 0;

    switch (op)
    {
    case SumOfProductsX1Y:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[0] - mean1) * (coord[i].y - mean2);
        break;
    case SumOfProductsX2Y:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[1] - mean1) * (coord[i].y - mean2);
        break;
    case SumofProductsX1X2:
        for (int i = idx; i < ARR_SIZE; i += BLOCK_SIZE)
            sum += (coord[i].xs[0] - mean1) * (coord[i].xs[1] - mean2);
        break;
    }

    __shared__ double r[BLOCK_SIZE];
    r[idx] = sum;
    __syncthreads();
    for (int size = BLOCK_SIZE / 2; size > 0; size /= 2)
    { // uniform
        if (idx < size)
            r[idx] += r[idx + size];
        __syncthreads();
    }
    if (idx == 0)
        *out = r[0];
}

double SumWithCuda(const std::vector<MultivariateCoordinate> &mc, Operation op)
{
    // Variables on CPU
    const MultivariateCoordinate *coord = &mc[0];
    double out[ARR_SIZE];
    int n = mc.size();

    // Variables on GPU
    MultivariateCoordinate *dev_coord;
    double *dev_out;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate memory on GPU
    cudaStatus = hipMalloc((void **)&dev_coord, n * sizeof(MultivariateCoordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_out, n * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coord, n * sizeof(MultivariateCoordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy input failed!");
        goto Clean;
    }

    // Launch kernel
    calculate_sum<<<1, BLOCK_SIZE>>>(dev_coord, dev_out, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from output to input
    cudaStatus = hipMemcpy(out, dev_out, n * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy output failed!");
        goto Clean;
    }

    return out[0];

Clean:
    hipFree(dev_coord);
    hipFree(dev_out);
}

double SumWithCuda(const std::vector<MultivariateCoordinate> &mc, double mean, Operation op)
{
    // Variables on CPU
    const MultivariateCoordinate *coord = &mc[0];
    double out[ARR_SIZE];
    int n = mc.size();

    // Variables on GPU
    MultivariateCoordinate *dev_coord;
    double *dev_out;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate memory on GPU
    cudaStatus = hipMalloc((void **)&dev_coord, n * sizeof(MultivariateCoordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_out, n * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coord, n * sizeof(MultivariateCoordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy input failed!");
        goto Clean;
    }

    // Launch kernel
    calculate_sum<<<1, BLOCK_SIZE>>>(dev_coord, dev_out, mean, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from output to input
    cudaStatus = hipMemcpy(out, dev_out, n * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy output failed!");
        goto Clean;
    }

    return out[0];

Clean:
    hipFree(dev_coord);
    hipFree(dev_out);
}

double SumWithCuda(const std::vector<MultivariateCoordinate> &mc, double mean1, double mean2, Operation op)
{
    // Variables on CPU
    const MultivariateCoordinate *coord = &mc[0];
    double out[ARR_SIZE];
    int n = mc.size();

    // Variables on GPU
    MultivariateCoordinate *dev_coord;
    double *dev_out;
    hipError_t cudaStatus;

    // Choose GPU
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Clean;
    }

    // Allocate memory on GPU
    cudaStatus = hipMalloc((void **)&dev_coord, n * sizeof(MultivariateCoordinate));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    cudaStatus = hipMalloc((void **)&dev_out, n * sizeof(double));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        goto Clean;
    }

    // Copy input data from CPU to GPU
    cudaStatus = hipMemcpy(dev_coord, coord, n * sizeof(MultivariateCoordinate), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy input failed!");
        goto Clean;
    }

    // Launch kernel
    calculate_sum<<<1, BLOCK_SIZE>>>(dev_coord, dev_out, mean1, mean2, op);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Clean;
    }

    // Wait for kernel to finish
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Clean;
    }

    // Copy output data from output to input
    cudaStatus = hipMemcpy(out, dev_out, n * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy output failed!");
        goto Clean;
    }

    return out[0];

Clean:
    hipFree(dev_coord);
    hipFree(dev_out);
}

std::tuple<std::pair<double, double>, double> CalculateGradientAndYIntercept(const std::vector<MultivariateCoordinate> &mc)
{
    Operation op;

    // Total number of coordinates
    int n = mc.size();

    // Calculate sum of x1, sum of x2 and sum of y
    op = SumOfX1;
    double sum_of_x1 = SumWithCuda(mc, op);
    op = SumOfX2;
    double sum_of_x2 = SumWithCuda(mc, op);
    op = SumOfY;
    double sum_of_y = SumWithCuda(mc, op);

    // Calculate mean
    double mean_x1 = sum_of_x1 / n;
    double mean_x2 = sum_of_x2 / n;
    double mean_y = sum_of_y / n;

    // Calculate sum of squares
    op = SumOfSquaresX1;
    double sum_of_squares_x1 = SumWithCuda(mc, mean_x1, op);
    op = SumOfSquaresX2;
    double sum_of_squares_x2 = SumWithCuda(mc, mean_x2, op);

    // Calculate sum of products
    op = SumOfProductsX1Y;
    double sum_of_products_x1_y = SumWithCuda(mc, mean_x1, mean_y, op);
    op = SumOfProductsX2Y;
    double sum_of_products_x2_y = SumWithCuda(mc, mean_x2, mean_y, op);
    op = SumofProductsX1X2;
    double sum_of_products_x1_x2 = SumWithCuda(mc, mean_x1, mean_x2, op);

    // Calculate b1, b2 and a
    double b1 = (sum_of_products_x1_y * sum_of_squares_x2 - sum_of_products_x1_x2 * sum_of_products_x2_y) / (sum_of_squares_x1 * sum_of_squares_x2 - sum_of_products_x1_x2 * sum_of_products_x1_x2);

    double b2 = (sum_of_products_x2_y * sum_of_squares_x1 - sum_of_products_x1_x2 * sum_of_products_x1_y) / (sum_of_squares_x1 * sum_of_squares_x2 - sum_of_products_x1_x2 * sum_of_products_x1_x2);

    double a = mean_y - (b1 * mean_x1) - (b2 * mean_x2);

    // Test return
    return std::make_tuple(std::make_pair(b1, b2), a);
}

// hipDeviceReset must be called before exiting in order for profiling and
// tracing tools such as Nsight and Visual Profiler to show complete traces.
void EndCuda()
{
    hipError_t cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");
    }
}